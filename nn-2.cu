#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//#include <time.h>

#include "nn-2.h"
#include "nn-2_cuda.cu"

// note cannot include one more time utils-2.c

typedef struct {
 
 	// weights init and bias is an issues
 
    int n_inputs;
    int n_hidden;
    int n_outputs;

    float *out_input;
    float *out_hidden;
    float *out_output;

    float *changes_input_hidden;
    float *changes_hidden_output;

    float *w_input_hidden;
    float *w_hidden_output;
} NeuralNet;

/* change Pattern from int to floating
typedef struct {
    int *result;
    int *data;
} Pattern;
*/

typedef struct {
    float *result;
    float *data;
} Patternf;


void buildLayer(float *arr, int n, float initial) {
    
    // why this a layer
    // we need layers per neutron layer ?
    
    int i=0;
    while(i < n){
    
        // why change array convention
        // can use arr[] 
        
        *arr = initial;
        arr++;
        i++;
    }
}

float* buildWeightsLayer(int outer_n, int inner_n, float seed) {

     // no bias
     // no allowance of different weights
     //    But if allow defeat the init purpose
     // it should be a 2 dim array
     // weights[inner_layer+1 outer layer] with bias

    int total = outer_n * inner_n;
    float *w = (float *)malloc(sizeof(float) * total);
	for(int i=0; i < total; i++) {
		if (seed == -1.00) { // not -1 ??
		  w[i] = ((float)rand()/(float)RAND_MAX);
		} else {
		  w[i] = seed;
		}
	}
    return w;
}

NeuralNet buildNeuralNet(int n_inputs, int n_outputs, int n_hidden) {

    // ok for simple to assume only 1 "layer" of hidden ... need concept extension though
    
    // per each patternf p as input_feeder[p]
    
    // input
    // input2hidden  - fwd: weights and bias
    // hidden 
    // hidden2hidden - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // hidden
    // hidden2output - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // output
    //.              - error calc (or in patternf)
    
    // here it use the idea of out_ but need bac
    //.    And also input just has out no bwd
    
    // per each patternf p as expected_output[p]
    
    // batch
    // regularisation
    // era
    // delta 
    // ...
    
    // absolute minimum model is 2i-2h-2h-2o and patternfs.  

    float *out_input = (float *)malloc(sizeof(float) * (n_inputs + 1)); // need 1 extra ? got bias
    
    float *out_hidden = (float *)malloc(sizeof(float) * n_hidden); // no 1 extra ? no bias
    
    float *out_output = (float *)malloc(sizeof(float) * n_outputs);

    buildLayer(out_input, n_inputs + 1, 1.0f);  // why plus 1 here ??
    
    buildLayer(out_hidden, n_hidden, 1.0f);
    buildLayer(out_output, n_outputs, 1.0f);

    // Build changes layer ? not sure what is this
    
    float *changes_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden, 0.0f);
    float *changes_hidden_output = buildWeightsLayer(n_hidden, n_outputs, 0.0f);

    // Build weight matrix
    
    float *w_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden,  -1.0f); // random
    float *w_hidden_output = buildWeightsLayer(n_hidden, n_outputs,  -1.0f); // random)

	w_input_hidden[0] = 0.15;
	w_input_hidden[1] = 0.20;
	w_input_hidden[2] = 0.35;
	w_input_hidden[3] = 0.25;
	w_input_hidden[4] = 0.30;
	w_input_hidden[5] = 0.35;
	
	w_hidden_output[0] = .40;
	w_hidden_output[1] = .45; // missing 0.60 no bias
	w_hidden_output[2] = .50;
	w_hidden_output[3] = .55; // missing 0.60 no bias


    NeuralNet nn;

    nn.n_inputs = n_inputs + 1;
    nn.n_outputs = n_outputs;
    nn.n_hidden = n_hidden;

    nn.out_input = out_input;
    nn.out_hidden = out_hidden;
    nn.out_output = out_output;

    nn.changes_input_hidden = changes_input_hidden;
    nn.changes_hidden_output = changes_hidden_output;

    nn.w_input_hidden = w_input_hidden;
    nn.w_hidden_output = w_hidden_output;

    return nn;
}

float dsigmoid(float y) {
    return 1.0 - pow(y,2.0f);
}

void print_nn(NeuralNet nn){
	printf("\n--nn start seems input +/-1 is for bias but only for input strangely --\n");
	
	int i; 
	printf("\n nn.n_inputs already plus 1: %d",    nn.n_inputs);
	printf("\n nn.n_hidden:            %d",    nn.n_hidden);
	printf("\n nn.n_outputs:           %d\n\n",nn.n_outputs);

	// no nn.n_inputs + 1
	for(i=0; i < (nn.n_inputs); i++)                {printf(" nn.out_input[%d]: %f\n",             i, nn.out_input[i]);};
	for(i=0; i < (nn.n_hidden); i++)                {printf(" nn.out_hidden[%d]: %f\n",            i, nn.out_hidden[i]);};
	for(i=0; i < (nn.n_outputs); i++)               {printf(" nn.out_output[%d]: %f\n",            i, nn.out_output[i]);};
	printf("\n");

	// no nn.n_inputs + 1
	for(i=0; i < ((nn.n_inputs)  *nn.n_hidden); i++)  {printf(" nn.changes_input_hidden[%d]: %f\n",  i, nn.changes_input_hidden[i]);};
	for(i=0; i < ((nn.n_hidden)  *nn.n_outputs); i++) {printf(" nn.changes_hidden_output[%d]: %f\n", i, nn.changes_hidden_output[i]);};
	printf("\n");

	drawMatrix(nn.changes_input_hidden,  nn.n_inputs, nn.n_hidden);
	printf("\n");
	drawMatrix(nn.changes_hidden_output, nn.n_hidden,   nn.n_outputs);
	printf("\n");

    // no nn.n_inputs + 1
	for(i=0; i < ((nn.n_inputs)  *nn.n_hidden); i++)  {printf(" nn.w_input_hidden[%d]: %f\n",        i, nn.w_input_hidden[i]);};
	for(i=0; i < ((nn.n_hidden)  *nn.n_outputs); i++) {printf(" nn.w_hidden_output[%d]: %f\n",       i, nn.w_hidden_output[i]);};
    printf("\n");
	
	drawMatrix(nn.w_input_hidden,  nn.n_inputs, nn.n_hidden);
	printf("\n");
	drawMatrix(nn.w_hidden_output, nn.n_hidden,   nn.n_outputs);
	printf("\n");
	
	printf("\n--nn end   --\n");
	
}

void update_patternf(Patternf patternf, NeuralNet nn) {

    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-a ***** LAYER UPDATE *****\n");
        print_nn(nn);
    }

    // Write inputs // mixing all 3 togethers
    int i;
    for(i=0; i < (nn.n_inputs -1); i++) {        // -1 here ... why??
        nn.out_input[i] = patternf.data[i];     // why pattern.data[i] here ??? here it will store of these data in out_input[i]
    }

    // Run parallel update and amend to use cuda 
    
    update_layer_CUDA(nn.out_input,  nn.out_hidden, nn.n_inputs, nn.n_hidden,  nn.w_input_hidden);
    update_layer_CUDA(nn.out_hidden, nn.out_output, nn.n_hidden, nn.n_outputs, nn.w_hidden_output);

    if (DEBUG | DEBUG2) {
        printf("\n DEBUG2-b ***** END LAYER UPDATE *****\n");
    }
}

float back_propagate_network(Patternf p, NeuralNet n) {

    // no parallel? No cuda?? Why not all in cuda once built???

    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-c ***** BACK PROPAGATE *****\n");
    }

    int i, j;
    float *output_delta = (float*)malloc(sizeof(float) * n.n_outputs);
    float *hidden_delta = (float*)malloc(sizeof(float) * n.n_hidden);


    // Calculate output delta
    for (i=0; i < n.n_outputs; i++) {
        float error = p.result[i] - n.out_output[i];
        output_delta[i] = dsigmoid(n.out_output[i]) * error;
    }


    // Calculate hidden delta
    for(i=0; i < n.n_hidden; i++) {
        float error = 0.0f;
        for (j=0; j < n.n_outputs; j++) {
            error += output_delta[j] * n.w_hidden_output[i * n.n_outputs + j];
        }
        hidden_delta[i] = dsigmoid(n.out_hidden[i]) * error;
    }

    // Set hidden-output weights
    setWeightsForLayers_CUDA(n.w_hidden_output, n.changes_hidden_output, output_delta, n.out_hidden, n.n_hidden, n.n_outputs);
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-d Hidden-Output weights\n");
        drawMatrix(n.w_hidden_output, n.n_outputs, n.n_hidden);
        _sleep(1);  // why need to sleep ?
    }

    setWeightsForLayers_CUDA(n.w_input_hidden, n.changes_input_hidden, hidden_delta, n.out_input, n.n_inputs, n.n_hidden);
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-e Input-Hidden weights\n");
        drawMatrix(n.w_input_hidden, n.n_hidden, n.n_inputs);
        _sleep(1);  // why need to sleep ?
    }

    // Calculate error
    float error = 0.0f;
    for (i=0; i < n.n_outputs; i++) {
        error = error + 0.5f * pow(p.result[i] - n.out_output[i], 2);
    }
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-f ***** Error for this patternf is: %f *****\n", error);
        _sleep(2); // why need to sleep ?
    }
    return error;
}


void train_network(Patternf *patternfs, int n_patternfs, int n_iterations, NeuralNet nn) {
  int i, j;
  for (i=0; i < n_iterations; i++) {
    float error = 0;
    for (j=0; j < n_patternfs; j++) {
       update_patternf(patternfs[j], nn);
       error += back_propagate_network(patternfs[j], nn);
    }
    if (i % 10 == 0 | i < 10) {
       printf("nn-2-235 Error for iter %d is: %-.5f\n", i, error);
       if (DEBUG | DEBUG2) _sleep(2);  // why need sleep ???
    }
  }
}

/*
Pattern makePatternSingleOutput(int *data, int result) {
    Pattern p;
    p.data = data;

    p.result = (int *)malloc(sizeof(int));
    p.result[0] = result;

    return p;
}
*/

Patternf makePatternfSingleOutput(float *data, float *result) {

    Patternf p;
    
    p.data = data;
    p.result = result;
    
    return p;
}

void printPatternf(Patternf p){
    int i;
    for(i=0; i < (NO_INPUT_NEURON);  i++) {printf("no:%d p.data:%f,",   i,p.data[i]);};
	for(i=0; i < (NO_OUTPUT_NEURON); i++) {printf(" no:%d p.result:%f", i,p.result[i]);};
}

int main (int argc, char *argv[]) {

	/* http://www.thegeekstuff.com/2013/01/c-argc-argv/ */

	/* Conversion string into int */
	int noOfRun;
	if (argc > 1)
		{noOfRun = atoi(argv[1]);
		printf("\nargv[1] in intger=%d\n\n",noOfRun);}


	printf("nn-2 253 ------------------ main() starting -------------------------------n");

    srand((unsigned)time(NULL));

    int n_inputs  = NO_INPUT_NEURON;   //2;  // shall use configuration ... ???
    int n_outputs = NO_OUTPUT_NEURON;  //1 -> 2;
	int n_hidden  = NO_HIDDEN_NEURON;  //4 -> 2;
	
	// assume 2 input neuron, 4 hidden neuron and 1 output neuron with bais
	
	// 00b -3x5-> xxxxb -5x1-> 1
	// 01b -3x5-> xxxxb -5x1-> 0
	// 10b -3x5-> xxxxb -5x1-> 1
	// 11b -3x5-> xxxxb -5x1-> 0
    
    // Build output layer
    NeuralNet nn = buildNeuralNet(n_inputs, n_outputs, n_hidden); 

    // Build training samples - real life shall use file ... 
    
    /*
    int _p1[] = {0,0};
    Pattern p1 = makePatternSingleOutput(_p1, 1); // memory issues and cannot use ({0,0}, 1) ?
    int _p2[] = {0,1};
    Pattern p2 = makePatternSingleOutput(_p2, 0);
    int _p3[] = {1,1};
    Pattern p3 = makePatternSingleOutput(_p3, 1);
    int _p4[] = {1,0};
    Pattern p4 = makePatternSingleOutput(_p4, 0);
    */
    
    // https://mattmazur.com/2015/03/17/a-step-by-step-backpropagation-example/
    // try the number there
    
    float _p1data[]   = {0.05,0.10};
    float _p1result[] = {0.01,0.99};
     
    Patternf p1 = makePatternfSingleOutput(_p1data, _p1result); // memory issues and cannot use ({0,0}, 1) ?
    
	
    Patternf patternfs[] = {p1}; // instead of p1,p2,p3,p4 just p1

	// printf(" ========= length of patterns[]: %lu\n", sizeof(patternfs) / sizeof(patternfs[0])); 
			// only in compile time i and i is int, f is floating, need lu or unsigned long 

	int leng_patternf = (int) (sizeof(patternfs) / sizeof(patternfs[0]));

	printf("\n ========= length of patternfs[]: %d\n", leng_patternf); 
	int i; 
	for(i=0; i < (leng_patternf); i++) 
		{printf(" patternfs[%d]: ", i); 
		 printPatternf(patternfs[i]);}
	printf("\n ========= No of run           : %d\n",  noOfRun); //NO_OF_RUN); 
	
    // Train the network
    train_network(patternfs, leng_patternf, noOfRun, nn); // NO_OF_RUN, nn);  
    	// 4 patterns  which is now calculated and run run 1000 times which now is NO_OF_RUN => noOfRun
    	// 4 and 2 meant 8 run e.g. 8 back prop ... 

	// Test the network (shall use different data but here it would be the same as it is logic)
	
    printf("\n\n nn-2-295 Testing the network mixing the build, validation and test idea due the data's nature\n"); 
    	// update pattern probably not train it I guess ?? 
    
    update_patternf(p1, nn);  // ?? p1 ... (0 0) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- patternf ???: nn.out_output[%d]: %f, p1.result[%d]: %f\n", i, nn.out_output[i], i, p1.result[i]);
    }
    
    /*
    update_pattern(p2, nn);  // ?? p2 ... (0 1) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 010: nn.out_output[i]: %f, p2.result[i]: %f\n", nn.out_output[i], p2.result[i]);
    }
    
    update_pattern(p3, nn);  // ?? p3 ... (1 1) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 111: nn.out_output[i]: %f, p3.result[i]: %f\n", nn.out_output[i], p3.result[i]);
    }
    
    update_pattern(p4, nn);  // ?? p4 ... (1 0) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 100: nn.out_output[i]: %f, p4.result[i]: %f\n", nn.out_output[i], p4.result[i]);
    }
    */
    
    hipDeviceReset();
    
    return 0;
}
