#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//#include <time.h>

#include "utils.c"
#include "parallel.cu"

#ifdef _WIN32
#include <Windows.h>
#else
#include <unistd.h>
#endif

// use this and then if there is -DDEBUG it would be set but if not then it is false!

#ifndef DEBUG
#define DEBUG false
#endif

#ifndef DEBUG2
#define DEBUG2 false
#endif


//#ifdef __APPLE__
//    #include <unistd.h>
//#else _WIN32
//    #include <windows.h>
//#endif


typedef struct 

    // weights init and bias is an issues

    int n_inputs;
    int n_hidden;
    int n_outputs;

    float *out_input;
    float *out_hidden;
    float *out_output;

    float *changes_input_hidden;
    float *changes_hidden_output;

    float *w_input_hidden;
    float *w_hidden_output;
} NeuralNet;

typedef struct {

    // not sure understand this

    int *result;
    int *data;
} Pattern;

void buildLayer(float *arr, int n, float initial) {
    
    // why this a layer
    // we need layers per neutron layer ?
    
    int i=0;
    while(i < n){
    
        // why change array convention
        // can use arr[] 
        
        *arr = initial;
        arr++;
        i++;
    }
}

float* buildWeightsLayer(int outer_n, int inner_n, float seed) {

     // no bias
     // no allowance of different weights
     //    But if allow defeat the init purpose
     // it should be a 2 dim array
     // weights[inner_layer+1 outer layer] with bias

    int total = outer_n * inner_n;
    float *w = (float *)malloc(sizeof(float) * total);
    for(int i=0; i < total; i++) {
        if (seed == -1) {
          w[i] = ((float)rand()/(float)RAND_MAX);
        } else {
          w[i] = seed;
        }
    }
    return w;
}

NeuralNet buildNeuralNet(int n_inputs, int n_outputs, int n_hidden) {

    // ok for simple to assume only 1 "layer" of hidden ... need concept extension though
    
    // per each pattern p as input_feeder[p]
    
    // input
    // input2hidden  - fwd: weights and bias
    // hidden 
    // hidden2hidden - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // hidden
    // hidden2output - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // output
    //.              - error calc (or in pattern)
    
    // per each pattern p as expected_output[p]
    
    // batch
    // regularisation
    // era
    // delta 
    // ...
    
    // absolute minimum model is 2i-2h-2h-2o and patterns.  

    float *out_input = (float *)malloc(sizeof(float) * (n_inputs + 1)); // need 1 extra ? got bias
    
    float *out_hidden = (float *)malloc(sizeof(float) * n_hidden); // no 1 extra ? no bias
    
    float *out_output = (float *)malloc(sizeof(float) * n_outputs);

    buildLayer(out_input, n_inputs + 1, 1.0f);  // why plus 1 here ??
    
    buildLayer(out_hidden, n_hidden, 1.0f);
    buildLayer(out_output, n_outputs, 1.0f);

    // Build changes layer ? not sure what is this
    float *changes_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden, 0.0f);
    
    float *changes_hidden_output = buildWeightsLayer(n_hidden, n_outputs, 0.0f);

    // Build weight matrix
    float *w_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden, -1.0f);
    float *w_hidden_output = buildWeightsLayer(n_hidden, n_outputs, -1.0f);

    NeuralNet nn;

    nn.n_inputs = n_inputs + 1;
    nn.n_outputs = n_outputs;
    nn.n_hidden = n_hidden;

    nn.out_input = out_input;
    nn.out_hidden = out_hidden;
    nn.out_output = out_output;

    nn.changes_input_hidden = changes_input_hidden;
    nn.changes_hidden_output = changes_hidden_output;

    nn.w_input_hidden = w_input_hidden;
    nn.w_hidden_output = w_hidden_output;

    return nn;
}

float dsigmoid(float y) {
    return 1.0 - pow(y,2.0f);
}

void update_pattern(Pattern pattern, NeuralNet nn) {

    if (DEBUG2) {
        printf("\n nn-1-118 ***** LAYER UPDATE *****\n");
    }

    // Write inputs
    int i;
    for(i=0; i < nn.n_inputs -1; i++) {
        nn.out_input[i] = pattern.data[i];
    }

    // Run parallel update
    update_layer(nn.out_input, nn.out_hidden, nn.n_inputs, nn.n_hidden, nn.w_input_hidden);
    update_layer(nn.out_hidden, nn.out_output, nn.n_hidden, nn.n_outputs, nn.w_hidden_output);

    if (DEBUG2) {
        printf("\n nn-2-132 ***** END LAYER UPDATE *****\n");
    }
}

float back_propagate_network(Pattern p, NeuralNet n) {

    if (DEBUG2) {
        printf("\n nn-3-139 ***** BACK PROPAGATE *****\n");
    }

    int i, j;
    float *output_delta = (float*)malloc(sizeof(float) * n.n_outputs);
    float *hidden_delta = (float*)malloc(sizeof(float) * n.n_hidden);


    // Calculate output delta
    for (i=0; i < n.n_outputs; i++) {
        float error = p.result[i] - n.out_output[i];
        output_delta[i] = dsigmoid(n.out_output[i]) * error;
    }


    // Calculate hidden delta
    for(i=0; i < n.n_hidden; i++) {
        float error = 0.0f;
        for (j=0; j < n.n_outputs; j++) {
            error += output_delta[j] * n.w_hidden_output[i * n.n_outputs + j];
        }
        hidden_delta[i] = dsigmoid(n.out_hidden[i]) * error;
    }

    // Set hidden-output weights
    setWeightsForLayers(n.w_hidden_output, n.changes_hidden_output, output_delta, n.out_hidden, n.n_hidden, n.n_outputs);
    if (DEBUG2) {
        printf("\n nn-4-166 Hidden-Output weights\n");
        drawMatrix(n.w_hidden_output, n.n_outputs, n.n_hidden);
        _sleep(1);  // why need to sleep ?
    }

    setWeightsForLayers(n.w_input_hidden, n.changes_input_hidden, hidden_delta, n.out_input, n.n_inputs, n.n_hidden);
    if (DEBUG2) {
        printf("\n nn-5-173 Input-Hidden weights\n");
        drawMatrix(n.w_input_hidden, n.n_hidden, n.n_inputs);
        _sleep(1);  // why need to sleep ?
    }

    // Calculate error
    float error = 0.0f;
    for (i=0; i < n.n_outputs; i++) {
        error = error + 0.5f * pow(p.result[i] - n.out_output[i], 2);
    }
    if (DEBUG2) {
        printf("\n nn-6-184 ***** Error for this pattern is: %f *****\n", error);
        _sleep(2); // why need to sleep ?
    }
    return error;
}


void train_network(Pattern *patterns, int n_patterns, int n_iterations, NeuralNet nn) {
  int i, j;
  for (i=0; i < n_iterations; i++) {
    float error = 0;
    for (j=0; j < n_patterns; j++) {
       update_pattern(patterns[j], nn);
       error += back_propagate_network(patterns[j], nn);
    }
    if (i % 10 == 0) {
       printf("nn-7-200 Error is: %-.5f\n", error);
       if (DEBUG2) _sleep(2);
    }
  }
}

Pattern makePatternSingleOutput(int *data, int result) {
    Pattern p;
    p.data = data;

    p.result = (int *)malloc(sizeof(int));
    p.result[0] = result;

    return p;
}

int main() {

	printf("nn-8 218 ------------------ starting -------------------------------n");

    srand((unsigned)time(NULL));

    int n_inputs = 2;
    int n_outputs = 1;
	int n_hidden = 4;
	
	// assume 2 input neuron, 4 hidden neuron and 1 output neuron with bais
	
	// 00b -3x5-> xxxxb -5x1-> 1
	// 01b -3x5-> xxxxb -5x1-> 0
	// 10b -3x5-> xxxxb -5x1-> 1
	// 11b -3x5-> xxxxb -5x1-> 0
    
    // Build output layer
    NeuralNet nn = buildNeuralNet(n_inputs, n_outputs, n_hidden); 

    // Build training samples
    int _p1[] = {0,0};
    Pattern p1 = makePatternSingleOutput(_p1, 1);
    int _p2[] = {0,1};
    Pattern p2 = makePatternSingleOutput(_p2, 0);
    int _p3[] = {1,1};
    Pattern p3 = makePatternSingleOutput(_p3, 1);
    int _p4[] = {1,0};
    Pattern p4 = makePatternSingleOutput(_p4, 0);

    Pattern patterns[] = {p1, p2, p3, p4}; // use 1,2,3,4 instead of 3,2,1,4 ...?

    // Train the network
    train_network(patterns, 4, 1000, nn);  // 4 patterns and run run 1000 times

    printf("\n\n nn-9-251 Testing the network\n"); // ?? why only update p2 ... (0 1) -> 0 
    update_pattern(p1, nn);  // ?? p2 ... (0 0) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- nn-10-254 Output: %f, expected 1 index: %i\n", nn.out_output[i], p2.result[i]);
    }
    update_pattern(p2, nn);  // ?? p2 ... (0 1) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- nn-10-258 Output: %f, expected 0 index: %i\n", nn.out_output[i], p2.result[i]);
    }
    update_pattern(p3, nn);  // ?? p2 ... (1 1) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- nn-10-262 Output: %f, expected 1 index: %i\n", nn.out_output[i], p2.result[i]);
    }
    update_pattern(p4, nn);  // ?? p2 ... (1 0) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- nn-10-266 Output: %f, expected 0 index: %i\n", nn.out_output[i], p2.result[i]);
    }
    hipDeviceReset();
    return 0;
}
