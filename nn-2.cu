#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//#include <time.h>

#include "nn-2.h"
#include "nn-2_cuda.cu"

// note cannot include one more time utils-2.c

typedef struct {
 
 	// weights init and bias is an issues
 
    int n_inputs;
    int n_hidden;
    int n_outputs;

    float *out_input;
    float *out_hidden;
    float *out_output;

    float *changes_input_hidden;
    float *changes_hidden_output;

    float *w_input_hidden;
    float *w_hidden_output;
} NeuralNet;

typedef struct {
    int *result;
    int *data;
} Pattern;

void buildLayer(float *arr, int n, float initial) {
    
    // why this a layer
    // we need layers per neutron layer ?
    
    int i=0;
    while(i < n){
    
        // why change array convention
        // can use arr[] 
        
        *arr = initial;
        arr++;
        i++;
    }
}

float* buildWeightsLayer(int outer_n, int inner_n, float seed) {

     // no bias
     // no allowance of different weights
     //    But if allow defeat the init purpose
     // it should be a 2 dim array
     // weights[inner_layer+1 outer layer] with bias

    int total = outer_n * inner_n;
    float *w = (float *)malloc(sizeof(float) * total);
    for(int i=0; i < total; i++) {
        if (seed == -1) {
          w[i] = ((float)rand()/(float)RAND_MAX);
        } else {
          w[i] = seed;
        }
    }
    return w;
}

NeuralNet buildNeuralNet(int n_inputs, int n_outputs, int n_hidden) {

    // ok for simple to assume only 1 "layer" of hidden ... need concept extension though
    
    // per each pattern p as input_feeder[p]
    
    // input
    // input2hidden  - fwd: weights and bias
    // hidden 
    // hidden2hidden - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // hidden
    // hidden2output - fwd: weights and bias
    //.              - bwd: delta (or hidden)
    // output
    //.              - error calc (or in pattern)
    
    // here it use the idea of out_ but need bac
    //.    And also input just has out no bwd
    
    // per each pattern p as expected_output[p]
    
    // batch
    // regularisation
    // era
    // delta 
    // ...
    
    // absolute minimum model is 2i-2h-2h-2o and patterns.  

    float *out_input = (float *)malloc(sizeof(float) * (n_inputs + 1)); // need 1 extra ? got bias
    
    float *out_hidden = (float *)malloc(sizeof(float) * n_hidden); // no 1 extra ? no bias
    
    float *out_output = (float *)malloc(sizeof(float) * n_outputs);

    buildLayer(out_input, n_inputs + 1, 1.0f);  // why plus 1 here ??
    
    buildLayer(out_hidden, n_hidden, 1.0f);
    buildLayer(out_output, n_outputs, 1.0f);

    // Build changes layer ? not sure what is this
    float *changes_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden, 0.0f);
    
    float *changes_hidden_output = buildWeightsLayer(n_hidden, n_outputs, 0.0f);

    // Build weight matrix
    float *w_input_hidden = buildWeightsLayer(n_inputs + 1, n_hidden, -1.0f);
    
    float *w_hidden_output = buildWeightsLayer(n_hidden, n_outputs, -1.0f);

    NeuralNet nn;

    nn.n_inputs = n_inputs + 1;
    nn.n_outputs = n_outputs;
    nn.n_hidden = n_hidden;

    nn.out_input = out_input;
    nn.out_hidden = out_hidden;
    nn.out_output = out_output;

    nn.changes_input_hidden = changes_input_hidden;
    nn.changes_hidden_output = changes_hidden_output;

    nn.w_input_hidden = w_input_hidden;
    nn.w_hidden_output = w_hidden_output;

    return nn;
}

float dsigmoid(float y) {
    return 1.0 - pow(y,2.0f);
}

void print_nn(NeuralNet nn){
	printf("\n--nn start seems input +/-1 is for bias but only for input strangely --\n");
	
	int i; 
	
	for(i=0; i < (nn.n_inputs+1); i++)                {printf(" nn.out_input[%d]: %f\n",             i, nn.out_input[i]);};
	for(i=0; i < (nn.n_hidden); i++)                  {printf(" nn.out_hidden[%d]: %f\n",            i, nn.out_hidden[i]);};
	for(i=0; i < (nn.n_outputs); i++)                 {printf(" nn.out_output[%d]: %f\n",            i, nn.out_output[i]);};
	printf("\n");

	for(i=0; i < ((nn.n_inputs+1)*nn.n_hidden); i++)  {printf(" nn.changes_input_hidden[%d]: %f\n",  i, nn.changes_input_hidden[i]);};
	for(i=0; i < ((nn.n_hidden)  *nn.n_outputs); i++) {printf(" nn.changes_hidden_output[%d]: %f\n", i, nn.changes_hidden_output[i]);};
	printf("\n");

	for(i=0; i < ((nn.n_inputs+1)*nn.n_hidden); i++)  {printf(" nn.w_input_hidden[%d]: %f\n",        i, nn.w_input_hidden[i]);};
	for(i=0; i < ((nn.n_hidden)  *nn.n_outputs); i++) {printf(" nn.w_hidden_output[%d]: %f\n",       i, nn.w_hidden_output[i]);};
    printf("\n");
		
	printf("\n--nn end   --\n");
	
}

void update_pattern(Pattern pattern, NeuralNet nn) {

    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-a ***** LAYER UPDATE *****\n");
        print_nn(nn);
    }

    // Write inputs // mixing all 3 togethers
    int i;
    for(i=0; i < (nn.n_inputs -1); i++) {        // -1 here ... why??
        nn.out_input[i] = pattern.data[i];     // why pattern.data[i] here ??? here it will store of these data in out_input[i]
    }

    // Run parallel update and amend to use cuda 
    update_layer_CUDA(nn.out_input, nn.out_hidden, nn.n_inputs, nn.n_hidden, nn.w_input_hidden);
    
    update_layer_CUDA(nn.out_hidden, nn.out_output, nn.n_hidden, nn.n_outputs, nn.w_hidden_output);

    if (DEBUG | DEBUG2) {
        printf("\n DEBUG2-b ***** END LAYER UPDATE *****\n");
    }
}

float back_propagate_network(Pattern p, NeuralNet n) {

    // no parallel? No cuda?? Why not all in cuda once built???

    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-c ***** BACK PROPAGATE *****\n");
    }

    int i, j;
    float *output_delta = (float*)malloc(sizeof(float) * n.n_outputs);
    float *hidden_delta = (float*)malloc(sizeof(float) * n.n_hidden);


    // Calculate output delta
    for (i=0; i < n.n_outputs; i++) {
        float error = p.result[i] - n.out_output[i];
        output_delta[i] = dsigmoid(n.out_output[i]) * error;
    }


    // Calculate hidden delta
    for(i=0; i < n.n_hidden; i++) {
        float error = 0.0f;
        for (j=0; j < n.n_outputs; j++) {
            error += output_delta[j] * n.w_hidden_output[i * n.n_outputs + j];
        }
        hidden_delta[i] = dsigmoid(n.out_hidden[i]) * error;
    }

    // Set hidden-output weights
    setWeightsForLayers_CUDA(n.w_hidden_output, n.changes_hidden_output, output_delta, n.out_hidden, n.n_hidden, n.n_outputs);
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-d Hidden-Output weights\n");
        drawMatrix(n.w_hidden_output, n.n_outputs, n.n_hidden);
        _sleep(1);  // why need to sleep ?
    }

    setWeightsForLayers_CUDA(n.w_input_hidden, n.changes_input_hidden, hidden_delta, n.out_input, n.n_inputs, n.n_hidden);
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-e Input-Hidden weights\n");
        drawMatrix(n.w_input_hidden, n.n_hidden, n.n_inputs);
        _sleep(1);  // why need to sleep ?
    }

    // Calculate error
    float error = 0.0f;
    for (i=0; i < n.n_outputs; i++) {
        error = error + 0.5f * pow(p.result[i] - n.out_output[i], 2);
    }
    if (DEBUG | DEBUG2c) {
        printf("\n DEBUG2-f ***** Error for this pattern is: %f *****\n", error);
        _sleep(2); // why need to sleep ?
    }
    return error;
}


void train_network(Pattern *patterns, int n_patterns, int n_iterations, NeuralNet nn) {
  int i, j;
  for (i=0; i < n_iterations; i++) {
    float error = 0;
    for (j=0; j < n_patterns; j++) {
       update_pattern(patterns[j], nn);
       error += back_propagate_network(patterns[j], nn);
    }
    if (i % 10 == 0 | i < 10) {
       printf("nn-2-235 Error for iter %d is: %-.5f\n", i, error);
       if (DEBUG | DEBUG2) _sleep(2);  // why need sleep ???
    }
  }
}

Pattern makePatternSingleOutput(int *data, int result) {
    Pattern p;
    p.data = data;

    p.result = (int *)malloc(sizeof(int));
    p.result[0] = result;

    return p;
}

int main() {

	printf("nn-2 253 ------------------ main() starting -------------------------------n");

    srand((unsigned)time(NULL));

    int n_inputs  = NO_INPUT_NEURON;   //2;  // shall use configuration ... ???
    int n_outputs = NO_OUTPUT_NEURON;  //1;
	int n_hidden  = NO_HIDDEN_NEURON;  //4 -> 2;
	
	// assume 2 input neuron, 4 hidden neuron and 1 output neuron with bais
	
	// 00b -3x5-> xxxxb -5x1-> 1
	// 01b -3x5-> xxxxb -5x1-> 0
	// 10b -3x5-> xxxxb -5x1-> 1
	// 11b -3x5-> xxxxb -5x1-> 0
    
    // Build output layer
    NeuralNet nn = buildNeuralNet(n_inputs, n_outputs, n_hidden); 

    // Build training samples - real life shall use file ... 
    int _p1[] = {0,0};
    Pattern p1 = makePatternSingleOutput(_p1, 1); // memory issues and cannot use ({0,0}, 1) ?
    int _p2[] = {0,1};
    Pattern p2 = makePatternSingleOutput(_p2, 0);
    int _p3[] = {1,1};
    Pattern p3 = makePatternSingleOutput(_p3, 1);
    int _p4[] = {1,0};
    Pattern p4 = makePatternSingleOutput(_p4, 0);

	
    Pattern patterns[] = {p1, p2, p3, p4}; // use 1,2,3,4 instead of 3,2,1,4 ...?

	// printf(" ========= length of patterns[]: %lu\n", sizeof(patterns) / sizeof(patterns[0])); 
			// only in compile time i and i is int, f is floating, need lu or unsigned long 

	int leng_pattern = (int) (sizeof(patterns) / sizeof(patterns[0]));

	printf("\n ========= length of patterns[]: %d\n", leng_pattern); 
	printf("\n ========= No of run           : %d\n", NO_OF_RUN); 
	
    // Train the network
    train_network(patterns, leng_pattern, NO_OF_RUN, nn);  
    	// 4 patterns  which is now calculated and run run 1000 times which now is NO_OF_RUN
    	// 4 and 2 meant 8 run e.g. 8 back prop ... 

	// Test the network (shall use different data but here it would be the same as it is logic)
	
    printf("\n\n nn-2-295 Testing the network mixing the build, validation and test idea due the data's nature\n"); 
    	// update pattern probably not train it I guess ?? 
    
    update_pattern(p1, nn);  // ?? p2 ... (0 0) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 001: nn.out_output[i]: %f, p2.result[i]: %i\n", nn.out_output[i], p2.result[i]);
    }
    
    update_pattern(p2, nn);  // ?? p2 ... (0 1) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 010: nn.out_output[i]: %f, p2.result[i]: %i\n", nn.out_output[i], p2.result[i]);
    }
    
    update_pattern(p3, nn);  // ?? p2 ... (1 1) -> 1
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 111: nn.out_output[i]: %f, p2.result[i]: %i\n", nn.out_output[i], p2.result[i]);
    }
    
    update_pattern(p4, nn);  // ?? p2 ... (1 0) -> 0
    for (int i=0; i < nn.n_outputs; i++) {
        printf(" ------------- pattern 100: nn.out_output[i]: %f, p2.result[i]: %i\n", nn.out_output[i], p2.result[i]);
    }
    
    hipDeviceReset();
    return 0;
}
