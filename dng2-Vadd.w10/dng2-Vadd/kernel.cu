#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

/* must use .cu otherwise .c and .cpp will send to host compiler and global would have issues */
/* under w10 */

__global__ void VectorAdd(int *a, int *b, int *c, int n) {
	int i = threadIdx.x;
	// no loop for (i = 0; i < n; ++i)
	if (i < n)
		c[i] = a[i] + b[i];
}

int main(int argc, char *argv[])
{

	int noOfRun;
	if (argc > 1)
	{
		noOfRun = atoi(argv[1]);
		printf("\nargv[1] in intger=%d\n\n", noOfRun);
	}

	// use SIZE here instead of noofRun

	int *a, *b, *c;

	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));

	int *d_a, *d_b, *d_c;

	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i + 1;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	VectorAdd << <1, SIZE >> >(d_a, d_b, d_c, SIZE);

	hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; ++i)
		printf("host = %d: a[%d] + b[%d] = %d + %d = c[%d] = %d\n", i, i, i, a[i], b[i], i, c[i]);

	/* you cannot directly address the gpu memory !!!
	for (int i = 0; i < 10; ++i)
	printf("device = %d: d_a[%d] + d_b[%d] = %d + %d = d_c[%d] = %d\n", i, i, i, d_a[i], d_b[i], i, d_c[i]); */

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// hipProfilerStop(); and _syncthreads(); and device level close ????

	return 0;
}

/*

#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

void VectorAdd(int *a, int *b, int *c, int n) {
int i;
for (i = 0; i < n; ++i)
c[i] = a[i] + b[i];
}

int main(int argc, char *argv[])
{

int noOfRun;
if (argc > 1)
{
noOfRun = atoi(argv[1]);
printf("\nargv[1] in intger=%d\n\n", noOfRun);
}

// use SIZE here instead of noofRun

int *a, *b, *c;

a = (int *)malloc(SIZE * sizeof(int));
b = (int *)malloc(SIZE * sizeof(int));
c = (int *)malloc(SIZE * sizeof(int));

for (int i = 0; i < SIZE; ++i)
{
a[i] = i;
b[i] = i + 1;
c[i] = 0;
}

VectorAdd(a, b, c, SIZE);

for (int i = 0; i < 10; ++i)
printf("%d: a[%d] + b[%d] = %d + %d = c[%d] = %d\n", i, i, i, a[i], b[i], i, c[i]);

free(a);
free(b);
free(c);

return 0;
}


*/

/*

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}

int main()
{
const int arraySize = 5;
const int a[arraySize] = { 1, 2, 3, 4, 5 };
const int b[arraySize] = { 10, 20, 30, 40, 50 };
int c[arraySize] = { 0 };

// Add vectors in parallel.
hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addWithCuda failed!");
return 1;
}

printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
c[0], c[1], c[2], c[3], c[4]);

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceReset failed!");
return 1;
}

return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
int *dev_a = 0;
int *dev_b = 0;
int *dev_c = 0;
hipError_t cudaStatus;

// Choose which GPU to run on, change this on a multi-GPU system.
cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
goto Error;
}

// Allocate GPU buffers for three vectors (two input, one output)    .
cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

// Copy input vectors from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

// Launch a kernel on the GPU with one thread for each element.
addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
goto Error;
}

// Copy output vector from GPU buffer to host memory.
cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

Error:
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);

return cudaStatus;
}
*/
