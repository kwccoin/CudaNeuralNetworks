#include "hip/hip_runtime.h"
#include <math.h>

// http://stackoverflow.com/questions/17076956/nvcc-linking-error-in-c-and-cuda-c-code
#include "nn-2_cuda.h"
#include "nn-2.h"

#include "utils-2.c"

#include "utils-2_cuda.cu"



/* ---------------- [[CUDA KERNELS]] ---------------- */

__global__ void updateWeights_CUDA(float *weights, float *changes, float *delta_outputs, float *inputs, int n_inputs, int n_outputs) {
    int width = n_outputs;
    int height = n_inputs;
    GlobalDim gd = getGlobalDim_CUDA(blockDim, blockIdx, threadIdx);

    if ((gd.x < width) && (gd.y < height)) {
        int idx = width * gd.y + gd.x;
        float change = delta_outputs[gd.x] * inputs[gd.y];
        
        weights[idx] += 0.5 * change + 0.5 * changes[idx]; // 0.5 is magic no. use two 1.0 instead
        changes[idx] = change;
    }

}

__global__ void mapStep_CUDA(float *inputs, float *matrix, float *buffer, int width, int height) {
    GlobalDim gd = getGlobalDim_CUDA(blockDim, blockIdx, threadIdx);

    if ((gd.x < width) && (gd.y < height)) {
        int idx = width * gd.y + gd.x;
        buffer[idx] = inputs[gd.y] * matrix[idx];
    }
}

__global__ void reduceStep_CUDA(float *input, float *output, int width, int height) {

    __shared__ float sharedMemory[WARP_SIZE * WARP_SIZE];

    // STEP 1: exclude all threads that do not depend from problem
    GlobalDim gd = getGlobalDim_CUDA(blockDim, blockIdx, threadIdx);


    if ((gd.x < width) && (gd.y < height)) {

        // STEP 2: Move to shared memory
        int gridId = gd.y * width + gd.x;
        int blockId = threadIdx.y * blockDim.x + threadIdx.x;
        sharedMemory[blockId] = input[gridId];
        __syncthreads();

        int n = (int)ceil((float)blockDim.y/2);
        while(n >= 1) {
            if (threadIdx.y < n) {

                if ((gd.y + n) < height) {
                    int firstIndex = blockId;
                    int secondIndex = blockDim.x * (threadIdx.y + n) + threadIdx.x;
                    sharedMemory[firstIndex] += sharedMemory[secondIndex];
                }
            }
            __syncthreads();
            if (n == 1) {
                break;
            } else {
                n = (int)ceil((float)n/2);
            }
        }
        __syncthreads();

        // STEP 3: Write back results
        if (threadIdx.y == 1) {
            output[blockIdx.y * width + gd.x] = sharedMemory[threadIdx.x];
        }
    }
}

/* ---------------- [[LAUNCH FUNCTIONS]] ---------------- */

void setWeightsForLayers_CUDA(float *weights, float *changes, float *delta_outputs, float *inputs, int n_inputs, int n_outputs) {

    // Copy to device memory
    int grid_size = n_inputs * n_outputs;
    float *weights_d = _copyHostDevice_CUDA(weights, grid_size);
    float *changes_d = _copyHostDevice_CUDA(changes, grid_size);
    float *delta_outputs_d = _copyHostDevice_CUDA(delta_outputs, n_outputs);
    float *inputs_d = _copyHostDevice_CUDA(inputs, n_inputs);

    // Define block structure
    dim3 block(WARP_SIZE, WARP_SIZE);
    dim3 grid = getGridBasedOnBlockSize_CUDA(n_outputs, n_inputs, WARP_SIZE);

    // RUN RUN RUN!
    updateWeights_CUDA<<<grid, block>>>(weights_d, changes_d, delta_outputs_d, inputs_d, n_inputs, n_outputs);

    // Copy back weights and momenutm
    weights = _copyDeviceHost_CUDA(weights_d, grid_size, weights);
    changes = _copyDeviceHost_CUDA(changes_d, grid_size, changes);
}

// at least consistent with Cuda ending

void update_layer_CUDA(float *src_layer, float *dst_layer, int src_n, int dst_n, float *weights) {
    dim3 block(WARP_SIZE, WARP_SIZE);

    float *src_layer_d, *weights_d, *buffer_d;
    int total = src_n * dst_n;
 
    // Allocate input in global memory
    src_layer_d = _copyHostDevice_CUDA(src_layer, src_n);
    weights_d = _copyHostDevice_CUDA(weights, total);
    hipMalloc((void**)&buffer_d, sizeof(float) * total);
 
    // Create block dimensions and run parallel update layer
    int gridX = (int)ceil((float)dst_n/WARP_SIZE);
    int gridY = (int)ceil((float)src_n/WARP_SIZE);
    dim3 grid(gridX, gridY);

    // RUN RUN RUN!
    if (DEBUGP) {
        printf("\n par-1-123 ***** Updating layer *****\n");

        printf("\n par-2-125 From drawMatrix(src_layer, src_n, 1\n");
        drawMatrix(src_layer, src_n, 1);

        printf("\nT par-3-128 o drawMatrix(weights, dst_n, src_n)\n");
        drawMatrix(weights, dst_n, src_n);
    }
    mapStep_CUDA<<<grid, block>>>(src_layer_d, weights_d, buffer_d, dst_n, src_n);

    // Set the current target to the input
    float *currentTarget = buffer_d;
    int currentHeight = src_n;

    while (currentHeight > 1) {

        // Calculate grid size
        int gridX = (int)ceil((float)dst_n/WARP_SIZE);
        int gridY = (int)ceil((float)currentHeight/WARP_SIZE);
        dim3 grid(gridX, gridY);

        // Allocate new buffer
        float *buffer_d;
        hipMalloc((void**)&buffer_d, sizeof(float) * (dst_n * gridY));
 
        // RUN RUN RUN!
        reduceStep_CUDA<<<grid, block>>>(currentTarget, buffer_d, dst_n, currentHeight);

        // Free old memory and keep track of the new one
        hipFree(currentTarget);
        currentHeight = grid.y;
        currentTarget = buffer_d;
    }

    dst_layer =_copyDeviceHost_CUDA(currentTarget, dst_n, dst_layer);
    for (int i=0; i < dst_n; i++) {
        dst_layer[i] = sigmoid(dst_layer[i]); // tanh(dst_layer[i]);  // just apply tanh???
    }

    if (DEBUGP) {
        printf("\n par-4-163 Result is drawMatrix(dst_layer, dst_n, 1) \n");
        drawMatrix(dst_layer, dst_n, 1);
        printf("\n par-5-165 ***** ENDED UPDATING LAYER *****\n");
        _sleep(1);
    }
}

