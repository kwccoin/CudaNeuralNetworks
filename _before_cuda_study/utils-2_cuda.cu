
#include <hip/hip_runtime.h>
// see http://stackoverflow.com/questions/14818084/what-is-the-proper-include-for-the-function-sleep-in-c

//#include <stdio.h>
//#include <stdlib.h>
//#include <time.h>

//# include "nn-2_cuda.h"


//#define WARP_SIZE 16
//#define DEBUG false
//#define DEBUG true
/*
// use this and then if there is -DDEBUG it would be set but if not then it is false!

#ifndef DEBUG
#define DEBUG false
#endif

#ifndef DEBUGU
#define DEBUGU false
#endif


#ifdef _WIN32
#include <Windows.h>
#else
#include <unistd.h>
#endif
*/

// should be 2 as cuda from non_cuda one

/* ---------------- [[HELPER FUNCTIONS FOR GLOBAL MEMORY]] ---------------- */

float *_copyHostDevice_CUDA(float *src, int src_size) {
    float *src_d;
    hipMalloc((void**)&src_d, sizeof(float) * src_size);
    hipMemcpy(src_d, src, sizeof(float) * src_size, hipMemcpyHostToDevice);
    return src_d;
}

float *_copyDeviceHost_CUDA(float *src, int src_size, float *dst=NULL) {
    float *target;
    if (dst == NULL) {
        target = (float*)malloc(sizeof(float) * src_size);
    } else {
        target = dst;
    }

    hipMemcpy(target, src, sizeof(float) * src_size, hipMemcpyDeviceToHost);
    return target;
}

/* ---------------- [[HELPER FUNCTIONS FOR TILING]] ---------------- */

typedef struct {
    int x;
    int y;
} GlobalDim;

__device__ GlobalDim getGlobalDim_CUDA(dim3 blockDim, dim3 blockIdx, dim3 threadIdx) {
    GlobalDim gd;
    gd.x = blockDim.x * blockIdx.x + threadIdx.x;
    gd.y = blockDim.y * blockIdx.y + threadIdx.y;
    return gd;
}

dim3 getGridBasedOnBlockSize_CUDA(int width, int height, int block_size) {
    int gridX = (int)ceil((float)width / block_size);
    int gridY = (int)ceil((float)height / block_size);
    return dim3(gridX, gridY);
}

